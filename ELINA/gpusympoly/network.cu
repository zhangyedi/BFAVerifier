#include "hip/hip_runtime.h"
/*
 *  GPUPoly library
 *  This source file is part of ELINA (ETH LIbrary for Numerical Analysis).
 *  ELINA is Copyright � 2020 Department of Computer Science, ETH Zurich
 *  This software is distributed under GNU Lesser General Public License Version 3.0.
 *  For more information, see the ELINA project website at:
 *  http://elina.ethz.ch
 *
 *  THE SOFTWARE IS PROVIDED "AS-IS" WITHOUT ANY WARRANTY OF ANY KIND, EITHER
 *  EXPRESS, IMPLIED OR STATUTORY, INCLUDING BUT NOT LIMITED TO ANY WARRANTY
 *  THAT THE SOFTWARE WILL CONFORM TO SPECIFICATIONS OR BE ERROR-FREE AND ANY
 *  IMPLIED WARRANTIES OF MERCHANTABILITY, FITNESS FOR A PARTICULAR PURPOSE,
 *  TITLE, OR NON-INFRINGEMENT.  IN NO EVENT SHALL ETH ZURICH BE LIABLE FOR ANY
 *  DAMAGES, INCLUDING BUT NOT LIMITED TO DIRECT, INDIRECT,
 *  SPECIAL OR CONSEQUENTIAL DAMAGES, ARISING OUT OF, RESULTING FROM, OR IN
 *  ANY WAY CONNECTED WITH THIS SOFTWARE (WHETHER OR NOT BASED UPON WARRANTY,
 *  CONTRACT, TORT OR OTHERWISE).
 */


 /*! \file src/network.cu
	  \brief Implementation of NeuralNetwork
	  \author Fran&ccedil;ois Serre

	  Implementation of the members of the class NeuralNetwork, defined in src/network.h.
  */

#include <iostream>
#include <fstream>
#include <string>
#include <algorithm>
#include <vector>
#include <assert.h>
#include <cmath>
#include <queue>
#include <thrust/fill.h>
#include "layers/input.h"
#include "filters.h"
#include "network.h"
#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include ""
using namespace std;


template <>
Vector<float>& NeuralNetwork::getConcreteBounds(int layer)
{
	return *concreteBoundsS[layer];
}

template <>
Vector<double>& NeuralNetwork::getConcreteBounds(int layer)
{
	return *concreteBoundsD[layer];
}

template <typename T>
__global__ void setFinal(T* A, int* rows, int label, size_t outputSize, size_t N)
{
	size_t i = threadIdx.x;
	rows[i] = i;
	for (int j = 0; j < outputSize; j++)
		if (j == i + (i >= label))
			A[i * N + j] = -1;
		else if (j == label)
			A[i * N + j] = 1;
		else
			A[i * N + j] = 0;
}


template <typename T>
void NeuralNetwork::evaluateAffine(Vector<T>& dest, const NeuronFilter<T>& al, int layer, bool up, bool sound, const std::shared_ptr<const Matrix<T>>& A, const std::shared_ptr<const Vector<T>>& b)
{
	// size of the expression
	int m = dest.size();
	// if it's bigger than current maxLayerSize, we change its value and deallocate existing annoyingNeuronLists
	if (m > maxLayerSize)
	{
		maxLayerSize = m;
		if (annoyingNeuronList)
		{
			hipFree(annoyingNeuronList);
			annoyingNeuronList = nullptr;
		}
		if (annoyingNeuronList2)
		{
			hipFree(annoyingNeuronList2);
			annoyingNeuronList2 = nullptr;
		}
	}
	if (!annoyingNeuronList)
		gpuErrchk(hipMalloc((void**)&annoyingNeuronList, maxLayerSize * sizeof(int)));
	if (!annoyingNeuronList2)
		gpuErrchk(hipMalloc((void**)&annoyingNeuronList2, maxLayerSize * sizeof(int)));

	int an = al.listCriticalNeurons(annoyingNeuronList, dest, annoyingNeurons);
	int maxNeurBP = (1 << 30) / (maxLayerSize * sizeof(Intv<T>));
	// std::cout<<"===evaluateAffine Begin=====\n";
	for (int start = 0; start < an; start += maxNeurBP)
	{
		int length = std::min(maxNeurBP, an - start);
		auto partialA = A ? std::make_shared<const Matrix<T>>(A->template selectRows<T> (length, annoyingNeuronList + start, false)) : nullptr;
		auto partialb = b ? std::make_shared<const Vector<T>>(b->template select<T>(length, annoyingNeuronList+start,false)) : nullptr;
		auto inExpr = AffineExpr<T>(length, layers[layer]->outputSize, layer, up, annoyingNeuronList + start,partialA,partialb,ConvShape(),sound);
		typename AffineExpr<T>::Queue exprs;
		exprs.push(inExpr);
		int nbEval = 0;
		while (!exprs.empty())
		{
			AffineExpr<T> tmp = exprs.top();
			// std::cout<< "tmp.layer = " << tmp.layer << "\n";
			// tmp.print();
			
			assert(tmp.sound == sound);
			exprs.pop();
			//assert(tmp.m == size);
			assert(tmp.n == layers[tmp.layer]->outputSize);
			if (exprs.empty())
			{
				// std::cout<< "Enter exprs.empty()\n" ;
				//concreteBounds[tmp.layer]->check();
				tmp.evaluateAndUpdate(dest, getConcreteBounds<T>(tmp.layer));
				//dest.check();
				nbEval++;

				if (nbEval > 1)
				{
					int an = al.listCriticalNeurons(annoyingNeuronList2, dest, annoyingNeurons, tmp.rows, tmp.m);
					if (an < tmp.m)
					{
						if (an == 0)
							return;
						tmp.selectRows(an, annoyingNeuronList2);
					}
				}

			}
			if (!exprs.empty() && exprs.top().layer == tmp.layer)
			{
				// std::cout << "Enter exprs.empty() && exprs.top().layer == tmp.layer\n";
				AffineExpr<T> tmp2 = exprs.top();
				exprs.pop();
				assert(tmp.sound == tmp2.sound);
				auto A = std::make_shared<Matrix<T>>();
				Matrix<T>::add(*A, *tmp.getA(), *tmp2.getA(),tmp.sound);
				std::shared_ptr<const Vector<T>> b;
				if (tmp.up)
					b = Vector<T> ::template add_dr<true> (tmp.b, tmp2.b);
				else
					b = Vector<T> ::template add_dr<false> (tmp.b, tmp2.b);
				ConvShape cs;
				if (tmp.cs && tmp2.cs)
				{
					if (
						tmp.cs.filters == tmp2.cs.filters &&
						tmp.cs.output_rows == tmp2.cs.output_rows &&
						tmp.cs.output_cols == tmp2.cs.output_cols &&
						tmp.cs.input_rows == tmp2.cs.input_rows &&
						tmp.cs.input_cols == tmp2.cs.input_cols &&
						tmp.cs.input_channels == tmp2.cs.input_channels
						)
					{
						if (tmp.cs.kernel_size_cols > tmp2.cs.kernel_size_cols)
							cs = tmp.cs;
						else
							cs = tmp2.cs;
					}
					else
					{
						std::cout << "Error merging:" << std::endl;
						tmp.cs.print();
						tmp2.cs.print();
						std::cout << std::endl;
					}

				}
				exprs.emplace(tmp.m, tmp.n, tmp.layer, tmp.up, tmp.rows, A, b, cs,tmp.sound);
			}
			else{
				// std::cout << "Enter else\n";
				layers[tmp.layer]->backSubstitute(exprs, tmp);
			}
		}
	}
}


template <int lgBlockSize>
__global__ void getSensitivityExprPrepareRes(int* rows, const int* oldRows, const size_t n)
{
	size_t idx = (blockIdx.x << lgBlockSize) + threadIdx.x;
	if (idx < n)
	{
		rows[idx] = oldRows[rows[idx]];
	}
}

template<typename T>
__global__ void makeIdMatrix(T* dest, size_t N, int outputSize, int start)
{
	int col = blockIdx.x * blockDim.x + threadIdx.x;
	int row = blockIdx.y;
	int output = row + start;
	if (col < outputSize)
		dest[row * N + col] = T(output == col);
}

template<typename T>
void NeuralNetwork::getSensitivity(T* const destA, T* const destb, int layer, bool up, bool sound, int m, const std::shared_ptr<const Matrix<T>>& A, const std::shared_ptr<const Vector<T>>& b)
{
	// if the number of expressions is bigger than current maxLayerSize, we change its value and deallocate existing annoyingNeuronLists
	if (m > maxLayerSize)
	{
		maxLayerSize = m;
		if (annoyingNeuronList)
		{
			hipFree(annoyingNeuronList);
			annoyingNeuronList = nullptr;
		}
		if (annoyingNeuronList2)
		{
			hipFree(annoyingNeuronList2);
			annoyingNeuronList2 = nullptr;
		}
	}

	// Allocate annoyingNeuronLists if needed
	if (!annoyingNeuronList)
		gpuErrchk(hipMalloc((void**)&annoyingNeuronList, maxLayerSize * sizeof(int)));
	if (!annoyingNeuronList2)
		gpuErrchk(hipMalloc((void**)&annoyingNeuronList2, maxLayerSize * sizeof(int)));

	// Maximal size of a chunk
	int maxNeurBP = (1 << 30) / (maxLayerSize * sizeof(Intv<T>));

	// Initialize annoyingNeuronList with a sequence from 0 to n-1
	thrust::sequence(thrust::device_pointer_cast<int>(annoyingNeuronList), thrust::device_pointer_cast<int>(annoyingNeuronList + m));

	// Initialize buffers for the resulting expression to be stored
	Matrix<T> resA;
	Vector<T> resb;
	
	// Split the expression into chunks, and process them one after the other	
	for (int start = 0; start < m; start += maxNeurBP)
	{
		// Size of current chunk
		int length = std::min(maxNeurBP, m - start);

		// Encapsulate the expression into an AffineExpr
		auto partialA = A ? std::make_shared<const Matrix<T>>(A->template selectRows<T>(length, annoyingNeuronList + start,false)) : nullptr;
		auto partialb = b ? std::make_shared<const Vector<T>>(b->template select<T>(length, annoyingNeuronList + start,false)) : nullptr;
		thrust::sequence(thrust::device_pointer_cast<int>(annoyingNeuronList + start), thrust::device_pointer_cast<int>(annoyingNeuronList + start + length)); // Reindex the chunk so that it starts from 0; this will make the final ordering easier, but we have to take into account start when copying in the buffer.
		auto inExpr = AffineExpr<T>(length, layers[layer]->outputSize, layer, up, annoyingNeuronList + start, partialA, partialb, ConvShape(), sound);

		// Creates a queue that will stack expressions to be added together (in case of residual networks or partial sums). The part with the highest layer number stays on top.
		typename AffineExpr<T>::Queue exprs;
		exprs.push(inExpr);

		// Loop while this queue is not empty
		while (!exprs.empty())
		{
			// Get the term on top
			AffineExpr<T> tmp = exprs.top();
			assert(tmp.sound == sound);
			exprs.pop();
			assert(tmp.n == layers[tmp.layer]->outputSize);

			// If the term is expressed in terms of the inputs, and we have only one term, we're done for this chunk.
			if (tmp.layer == 0 && exprs.empty())
			{
				// Prepare the output in the correct format, and reorders rows
				if (!tmp.A)
				{
					resA.reshape(tmp.m, tmp.n, sound);
					const int blockSize = 256;
					dim3 block(blockSize, 1, 1);
					dim3 grid((tmp.n + blockSize - 1) / blockSize, tmp.m, 1);
					if (sound)
						makeIdMatrix<Intv<T>> << <grid, block >> > (resA, resA.pitch(), tmp.n, start);
					else
						makeIdMatrix<T> << <grid, block >> > (resA, resA.pitch(), tmp.n, start);
					gpuErrchk(hipPeekAtLastError());
					gpuErrchk(hipDeviceSynchronize());
				}
				else
					resA = tmp.A->template selectRows<T>(length, tmp.rows, sound);
				if (!tmp.b)
				{
					resb.resize(length, sound);
					resb.zeroFill();
				}
				else
					resb = tmp.b->template select<T>(length, tmp.rows, sound);
				assert(resA.interval() == sound);
				assert(resb.interval() == sound);
				hipMemcpy2D(
					destA + start * tmp.n * sizeof(T) * (1 + sound),
					tmp.n * sizeof(T) * (1+sound),
					resA.data(), resA.pitchBytes(),
					tmp.n * sizeof(T) * (1+sound), tmp.m,
					hipMemcpyDeviceToHost);
				hipMemcpy(destb + start*sizeof(T)*(1+sound),
					resb.data(),
					tmp.m * sizeof(T) * (1 + sound),
					hipMemcpyDeviceToHost);
			}
			if (!exprs.empty() && exprs.top().layer == tmp.layer)
			{
				AffineExpr<T> tmp2 = exprs.top();
				exprs.pop();
				assert(tmp.sound == tmp2.sound);
				auto A = std::make_shared<Matrix<T>>();
				Matrix<T>::add(*A, *tmp.getA(), *tmp2.getA(), tmp.sound);
				std::shared_ptr<const Vector<T>> b;
				if (tmp.up)
					b = Vector<T> ::template add_dr<true>(tmp.b, tmp2.b);
				else
					b = Vector<T> ::template add_dr<false>(tmp.b, tmp2.b);
				ConvShape cs;
				if (tmp.cs && tmp2.cs)
				{
					if (
						tmp.cs.filters == tmp2.cs.filters &&
						tmp.cs.output_rows == tmp2.cs.output_rows &&
						tmp.cs.output_cols == tmp2.cs.output_cols &&
						tmp.cs.input_rows == tmp2.cs.input_rows &&
						tmp.cs.input_cols == tmp2.cs.input_cols &&
						tmp.cs.input_channels == tmp2.cs.input_channels
						)
					{
						if (tmp.cs.kernel_size_cols > tmp2.cs.kernel_size_cols)
							cs = tmp.cs;
						else
							cs = tmp2.cs;
					}
					else
					{
						std::cout << "Error merging:" << std::endl;
						tmp.cs.print();
						tmp2.cs.print();
						std::cout << std::endl;
					}

				}
				exprs.emplace(tmp.m, tmp.n, tmp.layer, tmp.up, tmp.rows, A, b, cs, tmp.sound);
			}
			else
				layers[tmp.layer]->backSubstitute(exprs, tmp);
		}
	}
}

template void NeuralNetwork::getSensitivity(double* const destA, double* const destb, int layer, bool up, bool sound, int m, const std::shared_ptr<const Matrix<double>>& A, const std::shared_ptr<const Vector<double>>& b);
template void NeuralNetwork::getSensitivity(float* const destA, float* const destb, int layer, bool up, bool sound, int m, const std::shared_ptr<const Matrix<float>>& A, const std::shared_ptr<const Vector<float>>& b);



NeuralNetwork::NeuralNetwork(const size_t inputSize) :
	layers(), maxLayerSize(0),
	concreteBoundsS(), concreteBoundsD(),
	annoyingNeuronList(nullptr),
	annoyingNeuronList2(nullptr)
{
	gpuErrchk(hipMalloc((void**)&annoyingNeurons, sizeof(int)));
	addLayer(new Input(*this,inputSize));
}

NeuralNetwork::~NeuralNetwork() {
	if (annoyingNeurons) // we were not moved
	{
		hipFree(annoyingNeurons);
		for (auto l : layers)
			delete l;
	}
	if (annoyingNeuronList)
		hipFree(annoyingNeuronList);
	if (annoyingNeuronList2)
		hipFree(annoyingNeuronList2);
}

int NeuralNetwork::addLayer(Layer* layer)
{
	assert(!annoyingNeuronList);
	size_t m = layer->outputSize;
	layers.push_back(layer);
	if (maxLayerSize < m)
		maxLayerSize = m;
	concreteBoundsS.push_back(std::make_shared<Vector<float>>(m, true));
	concreteBoundsD.push_back(std::make_shared<Vector<double>>(m, true));
	return layers.size() - 1;
}

template <typename T>
bool NeuralNetwork::run(const Vector<T>& input, const int label, bool sound)
{
	if (!annoyingNeuronList)
		gpuErrchk(hipMalloc((void**)&annoyingNeuronList, maxLayerSize * sizeof(int)));
	if (!annoyingNeuronList2)
		gpuErrchk(hipMalloc((void**)&annoyingNeuronList2, maxLayerSize * sizeof(int)));

	size_t outputSize = layers.back()->outputSize; // size of the output of the neural network
	getConcreteBounds<T>(0) = input;


	// creates an additional "layer" to check if the neuron "label" is the bigger one
	auto finalA = std::make_shared<Matrix<T>>(outputSize - 1, outputSize, false);
	setFinal<T> << <1, outputSize - 1 >> > (*finalA, annoyingNeuronList, label, outputSize, finalA->pitch());

	Vector<T> res;
	// Vector<T> test;
	
	// creates another "layer" to examine the output of the neural network
	// std::vector<std::vector<T>> data0 = std::vector<std::vector<T>>(outputSize,std::vector<T>(outputSize,0));
	// for(int i = 0; i < outputSize; i++) data0[i][i] = 1;
	// auto finalB = std::make_shared<Matrix<T>>(data0);

	

	// finalA->mvm(res, getConcreteBounds<T>(layers.size() - 1));
	// finalB->mvm(test, getConcreteBounds<T>(layers.size() - 1));
	
	// std::cout<<"Final A\n"; finalA->print();  res.print();
	// evaluateAffine<T>(res, AlwaysKeep<T>(), layers.size() - 1, false, sound, finalA);
	// evaluateAffine<T>(test, AlwaysKeep<T>(), layers.size() - 1, false, sound, finalB);
	// if (res.isPositive())
	// 	return true;


	// computes the coefficients for each layer
	for (int p = 1; p < layers.size(); p++){
		layers[p]->eval(getConcreteBounds<T>(p), sound, true);
		// getConcreteBounds<T>(p).print();
	}
	
	finalA->mvm(res, getConcreteBounds<T>(layers.size() - 1));
	// evaluateAffine<T>(res, AlwaysKeep<T>(), layers.size() - 1, true, sound, finalA);
	evaluateAffine<T>(res, AlwaysKeep<T>(), layers.size() - 1, false, sound, finalA);
	
	// finalB->mvm(test, getConcreteBounds<T>(layers.size() - 1));
	// evaluateAffine<T>(test, AlwaysKeep<T>(), layers.size() - 1, false, sound, finalB);
	// evaluateAffine<T>(test, AlwaysKeep<T>(), layers.size() - 1, true, sound, finalB);

	// printf("test="); test.print();
	// printf("res="); res.print();
	
	return res.isPositive();
}



template bool NeuralNetwork::run(const Vector<float>& input, const int label, bool sound);
template bool NeuralNetwork::run(const Vector<double>& input, const int label, bool sound);
